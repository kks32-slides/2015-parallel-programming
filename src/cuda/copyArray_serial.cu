#include "hip/hip_runtime.h"
#include <stdio.h>

// Define constants for array and thread block dimensions
// You don't need to change these
#define ARRAY_ELEMENTS 10
#define BLOCK_SIZE 10

// Stage 3: Implement the kernel
__global__ void copyArray(int *arraySrc_d, int *arrayDest_d)
{
    // There's only a single thread block in this example
    // so we can just use the thread index to access the array
	
    // To be implemented	
}


int main( int argc, char** argv) 
{
    // pointer for host memory
    int *arraySrc_h, *arrayDest_h;

    // pointer for device memory
    int *arraySrc_d, *arrayDest_d;

    size_t memSize = ARRAY_ELEMENTS * sizeof(int);
    arraySrc_h  = (int *) malloc(memSize);
    arrayDest_h = (int *) malloc(memSize);
    if (arraySrc_h == NULL || arrayDest_h == NULL){
        printf("\nError in host memory allocation. Exiting");
        exit(1);
    }

    // Stage 1: Allocate device memory
    // To be implemented	
    
    // Define Grid and Block dimensions - in this example there's only one block
    dim3 dimGrid(1);
    dim3 dimBlock(BLOCK_SIZE);

    //Set host source array with some test values
    for (int j = 0; j < ARRAY_ELEMENTS; j++)
    {
        arraySrc_h[j] = j;
    }

    //Stage 2: copy the source array to the device
    // To be implemented	

    //Stage 4: Call the kernel function
    // To be implemented	

    // block until the device has completed
    hipDeviceSynchronize();

    //Stage 5: Copy arrayDest_d back to the host arrayDest_h
    // To be implemented	

    // Verify the data was copied correctly
    for (int j = 0; j < ARRAY_ELEMENTS; j++)
    {
        if (arrayDest_h[j] != arraySrc_h[j])
        {
            printf("\nERROR: Destination array is not equal to Source array\n");
            exit(-1);
        }
    }


    // free device memory
    hipFree(arraySrc_d);
    hipFree(arrayDest_d);

    // free host memory
    free(arraySrc_h);
    free(arrayDest_h);

    printf("Correct!\n");

    return 0;
}



