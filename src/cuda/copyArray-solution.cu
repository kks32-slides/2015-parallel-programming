#include "hip/hip_runtime.h"
#include <stdio.h>

// Define constants for array and thread block dimensions
// You don't need to change these
#define ARRAY_ELEMENTS 10
#define BLOCK_SIZE 10

// Stage 3: Implement the kernel
__global__ void copyArray(int *arraySrc_d, int *arrayDest_d)
{
    // There's only a single thread block in this example
    // so we can just use the thread index to access the array
    if (threadIdx.x < ARRAY_ELEMENTS)
        arrayDest_d[threadIdx.x] = arraySrc_d[threadIdx.x];
}


int main( int argc, char** argv) 
{
    // pointer for host memory
    int *arraySrc_h, *arrayDest_h;

    // pointer for device memory
    int *arraySrc_d, *arrayDest_d;

    size_t memSize = ARRAY_ELEMENTS * sizeof(int);
    arraySrc_h  = (int *) malloc(memSize);
    arrayDest_h = (int *) malloc(memSize);
    if (arraySrc_h == NULL || arrayDest_h == NULL){
        printf("\nError in host memory allocation. Exiting");
        exit(1);
    }

    // Stage 1: Allocate device memory
    hipMalloc(&arraySrc_d, memSize);
    hipMalloc(&arrayDest_d, memSize);
    
    // Define Grid and Block dimensions - in this example there's only one block
    dim3 dimGrid(1);
    dim3 dimBlock(BLOCK_SIZE);

    //Set host source array with some test values
    for (int j = 0; j < ARRAY_ELEMENTS; j++)
    {
        arraySrc_h[j] = j;
    }

    //Stage 2: copy the source array to the device
    hipMemcpy(arraySrc_d, arraySrc_h, memSize, hipMemcpyHostToDevice);    

    //Stage 4: Call the kernel function
    copyArray<<<dimGrid, dimBlock>>>(arraySrc_d, arrayDest_d);

    // block until the device has completed
    hipDeviceSynchronize();

    //Stage 5: Copy arrayDest_d back to the host arrayDest_h
    hipMemcpy(arrayDest_h, arrayDest_d, memSize, hipMemcpyDeviceToHost);

    // Verify the data was copied correctly
    for (int j = 0; j < ARRAY_ELEMENTS; j++)
    {
        if (arrayDest_h[j] != arraySrc_h[j])
        {
            printf("\nERROR: Destination array is not equal to Source array\n");
            exit(-1);
        }
    }


    // free device memory
    hipFree(arraySrc_d);
    hipFree(arrayDest_d);

    // free host memory
    free(arraySrc_h);
    free(arrayDest_h);

    printf("Correct!\n");

    return 0;
}



